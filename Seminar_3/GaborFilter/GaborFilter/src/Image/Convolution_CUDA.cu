#include "hip/hip_runtime.h"
#include "Image/Convolution_CUDA.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <opencv2/opencv.hpp>

#define BLOCK_SIZE      16

#define GPU_ERROR_CHECK(ans) { GpuAssert((ans), __FILE__, __LINE__); }
inline void GpuAssert(const hipError_t code, const char *file, const int32_t line, const bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr,
                "GpuAssert: %s %s %d\n",
                hipGetErrorString(code),
                file,
                line);

        if (abort) {
            exit(code);
        }
    }
}

__global__
void CUDAKernel(cv::cuda::PtrStepSz<float> image, cv::cuda::PtrStepSz<float> output, cv::cuda::PtrStepSz<float> kernel) {
    const auto pixelRow = threadIdx.y + blockIdx.y * blockDim.y;
    const auto pixelColumn = threadIdx.x + blockIdx.x * blockDim.x;

    if (pixelRow >= image.rows || pixelColumn >= image.cols) {
        return;
    }

    const auto kernelCenterRow = (kernel.rows - 1) / 2;
    const auto kernelCenterColumn = (kernel.cols - 1) / 2;

    auto sum = 0.0F;
    for (auto kernelRow = -kernelCenterRow; kernelRow <= kernelCenterRow; kernelRow++) {
        for (auto kernelColumn = -kernelCenterColumn; kernelColumn <= kernelCenterColumn; kernelColumn++) {
            if (pixelRow + kernelRow <= 0 ||
                pixelRow + kernelRow >= image.rows ||
                pixelColumn + kernelColumn <= 0 ||
                pixelColumn + kernelColumn >= image.cols) {
                continue;
            }

            sum += kernel.ptr(kernelRow + kernelCenterRow)[kernelColumn + kernelCenterColumn] *
                   image.ptr(pixelRow + kernelRow)[pixelColumn + kernelColumn];
        }

    }

    output.ptr(pixelRow)[pixelColumn] = sum;
}

cv::Mat Convolution::CUDA(cv::Mat image, cv::Mat kernel) {
    auto output = image.clone();

    cv::cuda::GpuMat image_d(image);
    cv::cuda::GpuMat kernel_d(kernel);
    cv::cuda::GpuMat output_d(output);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(ceil((float)image.cols / threadsPerBlock.x), ceil((float)image.rows / threadsPerBlock.y));

    CUDAKernel <<<numBlocks, threadsPerBlock>>>(image_d, output_d, kernel_d);
    GPU_ERROR_CHECK(hipPeekAtLastError());
    GPU_ERROR_CHECK(hipDeviceSynchronize());

    output_d.download(output);

    image_d.release();
    kernel_d.release();
    output_d.release();

    return output;
}
