#include "hip/hip_runtime.h"
#include "Image/Convolution.hpp"

#ifdef __HIPCC__
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <opencv2/core/cuda.hpp>
#include <opencv2/core/cuda_types.hpp>
#endif

#include <opencv2/core/mat.hpp>
#include <opencv2/core/types.hpp>

#include <cstdint>
#include <cstdio>

#ifdef __HIPCC__

#define BLOCK_SIZE      16
#define GPU_ERROR_CHECK(ans) { GpuAssert((ans), __FILE__, __LINE__); }

inline void GpuAssert(const hipError_t code,
                      const char *file,
                      const int32_t line) {
    if (code == hipSuccess) {
        return;
    }

    fprintf(stderr,
            "GpuAssert: %s %s %d\n",
            hipGetErrorString(code),
            file,
            line);

    exit(code);
}

__global__
void CudaConvolve(cv::cuda::PtrStepSz<float> image,
                  cv::cuda::PtrStepSz<float> output,
                  cv::cuda::PtrStepSz<float> kernel) {
    const auto pixelRow = threadIdx.y + blockIdx.y * blockDim.y;
    const auto pixelColumn = threadIdx.x + blockIdx.x * blockDim.x;

    if (pixelRow >= image.rows || pixelColumn >= image.cols) {
        return;
    }

    const auto kernelCenterRow = (kernel.rows - 1) / 2;
    const auto kernelCenterColumn = (kernel.cols - 1) / 2;

    auto sum = 0.0F;
    for (auto kernelRow = -kernelCenterRow; kernelRow <= kernelCenterRow; kernelRow++) {
        for (auto kernelColumn = -kernelCenterColumn; kernelColumn <= kernelCenterColumn; kernelColumn++) {
            if (pixelRow + kernelRow <= 0 ||
                pixelRow + kernelRow >= image.rows ||
                pixelColumn + kernelColumn <= 0 ||
                pixelColumn + kernelColumn >= image.cols) {
                continue;
            }

            sum += kernel.ptr(kernelRow + kernelCenterRow)[kernelColumn + kernelCenterColumn] *
                   image.ptr(pixelRow + kernelRow)[pixelColumn + kernelColumn];
        }

    }

    output.ptr(pixelRow)[pixelColumn] = sum;
}

cv::Mat Convolution::Cuda(const cv::Mat &image, const cv::Mat &kernel) {
    auto output = image.clone();

    cv::cuda::GpuMat image_d(image);
    cv::cuda::GpuMat kernel_d(kernel);
    cv::cuda::GpuMat output_d(output);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(ceil((float)image.cols / threadsPerBlock.x), ceil((float)image.rows / threadsPerBlock.y));

    CudaConvolve<<<numBlocks, threadsPerBlock>>>(image_d, output_d, kernel_d);
    GPU_ERROR_CHECK(hipPeekAtLastError());
    GPU_ERROR_CHECK(hipDeviceSynchronize());

    output_d.download(output);

    image_d.release();
    kernel_d.release();
    output_d.release();

    return output;
}

#else
#pragma message ("CUDA-NOT-SUPPORTED!")
cv::Mat Convolution::Cuda(const cv::Mat &image, const cv::Mat &kernel) {
    return cv::Mat(image.rows, image.cols, CV_32F, cv::Scalar(0, 0, 0));
}
#endif
