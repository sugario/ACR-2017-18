#include "hip/hip_runtime.h"
#include "Image/Convolution.hpp"

#ifdef __HIPCC__
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <opencv2/core/cuda.hpp>
#include <opencv2/core/cuda_types.hpp>
#endif

#include <opencv2/core/mat.hpp>
#include <opencv2/core/types.hpp>

#include <cstdint>
#include <cstdio>

#ifdef __HIPCC__
#define BLOCK_SIZE      16U
#define GPU_ERROR_CHECK(ans) { GpuAssert((ans), __FILE__, __LINE__); }
inline void GpuAssert(const hipError_t code,
                      const char *file,
                      const int32_t line) {
    if (code == hipSuccess) {
        return;
    }

    fprintf(stderr,
            "GpuAssert: %s %s %d\n",
            hipGetErrorString(code),
            file,
            line);

    exit(code);
}

__global__
void CudaConvolve(const cv::cuda::PtrStepSz<float> image,
                  cv::cuda::PtrStepSz<float> output,
                  const cv::cuda::PtrStepSz<float> kernel) {
    const auto pixelRow = threadIdx.y + blockIdx.y * blockDim.y;
    const auto pixelColumn = threadIdx.x + blockIdx.x * blockDim.x;

    if (pixelRow >= image.rows || pixelColumn >= image.cols) {
        return;
    }

    const auto kernelCenterRow = (kernel.rows - 1) / 2;
    const auto kernelCenterColumn = (kernel.cols - 1) / 2;

    auto sum = 0.0F;
    for (auto kernelRow = -kernelCenterRow; kernelRow <= kernelCenterRow; kernelRow++) {
        for (auto kernelColumn = -kernelCenterColumn; kernelColumn <= kernelCenterColumn; kernelColumn++) {
            if (pixelRow + kernelRow <= 0 ||
                pixelRow + kernelRow >= image.rows ||
                pixelColumn + kernelColumn <= 0 ||
                pixelColumn + kernelColumn >= image.cols) {
                continue;
            }

            sum += kernel.ptr(kernelRow + kernelCenterRow)[kernelColumn + kernelCenterColumn] *
                   image.ptr(pixelRow + kernelRow)[pixelColumn + kernelColumn];
        }
    }

    output.ptr(pixelRow)[pixelColumn] = sum;
}

cv::Mat convolution::Cuda(const cv::Mat &image, const cv::Mat &kernel) {
    cv::cuda::GpuMat output_d(image.clone());

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(static_cast<uint32_t>(std::ceil(static_cast<float>(image.cols) / threadsPerBlock.x)),
                   static_cast<uint32_t>(std::ceil(static_cast<float>(image.rows) / threadsPerBlock.y)));

    CudaConvolve<<<numBlocks, threadsPerBlock>>>(cv::cuda::GpuMat(image),
                                                 output_d,
                                                 cv::cuda::GpuMat(kernel));
    GPU_ERROR_CHECK(hipPeekAtLastError());
    GPU_ERROR_CHECK(hipDeviceSynchronize());

    cv::Mat output;
    output_d.download(output);
    output_d.release();

    return output;
}
#else
#pragma message("CUDA-NOT-SUPPORTED!")
cv::Mat convolution::Cuda(const cv::Mat &image, const cv::Mat &kernel) {
    return cv::Mat(image.rows, image.cols, CV_32F, cv::Scalar(0, 0, 0));
}
#endif
